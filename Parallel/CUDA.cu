#include "hip/hip_runtime.h"
#include <stdio.h>     
#include <stdlib.h> 
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "tempo.h"

void mostrar(int *p, int capa);

__global__ void ordenarPilhaCres(int *p, int *pAux0, int *pAux1, int topo, int NT){	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int midx = idx*NT;
	int topoA0 = (midx)-1;
	int topoA1 = (midx)-1;
	topo = (topo*idx+NT)-1;
	if (topo != (midx)-1) {
            int maior, temp;
            while (topo != (midx)-1) {
                maior = p[topo];
		topo--;
                while (topo != (midx)-1) {	
                    temp = p[topo];
		    topo--;
                    if (temp > maior) {
			topoA0++;
			pAux0[topoA0] = maior;
                        maior = temp;
                    } else {
                        topoA0++;
			pAux0[topoA0] = temp;
                    }
                }
                topoA1++;
		pAux1[topoA1] = maior;
                while (topoA0 != (midx)-1) {
			topo++;
			p[topo] = pAux0[topoA0];
   			topoA0--;
                }
            }
            while (topoA1 != (midx)-1) {
			topo++;
			p[topo] = pAux1[topoA1];
   			topoA1--;
            }
        }
}

int main(int argc,char *argv[]){
	if(argc != 3){
		 printf("NOME DO EXECUTAVEL + TAMANHO DA PILHA + NUMERO DE THREADS \n");
		 return 1;
	}

	int TAMVETOR, NUMTHREAD, TAMMENORES;
	TAMVETOR = atoi(argv[1]);
	NUMTHREAD = atoi(argv[2]);
	TAMMENORES = TAMVETOR/NUMTHREAD;
	int gridSize, blockSize;
	blockSize = NUMTHREAD/2;
	gridSize = 2;


	if(NUMTHREAD>1000){
		printf(" MAX 1000 THREADS  \n");
		return 1;
	}
	if(NUMTHREAD%2!=0 || TAMVETOR%2!=0){
		printf(" COISEIA UM NUMERO PAR DE THREADS E TAMANHO DA PILHA  \n");
		return 1;
	}
	double a1  = NUMTHREAD,a2 = TAMVETOR;
	if((a2/a1) != TAMVETOR/NUMTHREAD){
		printf(" NÃO NÃO NÃO NÃO NÃO NÃO NÃO NÃO NÃO NÃO NÃO NÃO NÃO NÃO \n");
		printf(" TAMVETOR/NUMTHREAD DEVE DAR UM NUMERO INTEIRO \n");
		return 1;
	}

	tempo1();

	srand(time(NULL));
	int i;

	int *h_Pilha;
	int *h_Topo;
	int *h_Min;

	int *d_PilhaAux0;
	int *d_PilhaAux1;
	int *d_Pilha;

	int TAMVETOR_bytes = (TAMVETOR)*sizeof(int);
	int NUMTHREAD_bytes = NUMTHREAD*sizeof(int);

	h_Pilha = (int *) malloc(TAMVETOR_bytes);
	h_Topo = (int *) malloc(NUMTHREAD_bytes);
	h_Min = (int *) malloc(NUMTHREAD_bytes);

	hipMalloc( (void**)&d_Pilha, TAMVETOR_bytes);
	hipMalloc( (void**)&d_PilhaAux0, TAMVETOR_bytes);
	hipMalloc( (void**)&d_PilhaAux1, TAMVETOR_bytes);


	for(i = 0; i < TAMVETOR; i++){
		h_Pilha[i] = (rand()%TAMVETOR);
	}

	hipMemcpy(d_Pilha, h_Pilha, TAMVETOR_bytes, hipMemcpyHostToDevice);

	ordenarPilhaCres<<<gridSize,blockSize>>>(d_Pilha, d_PilhaAux0, d_PilhaAux1, TAMMENORES, TAMMENORES);

	hipMemcpy(h_Pilha, d_Pilha, TAMVETOR_bytes, hipMemcpyDeviceToHost);

	for(i = 0; i < NUMTHREAD; i++){
		h_Topo[i] = ((TAMMENORES)*(i+1))-1;
		h_Min[i] = (TAMMENORES*i);
	}

	int maior = -1, u, temp;
	int *pilhaFinal;
	pilhaFinal = (int *) malloc(TAMVETOR_bytes);
	int topoFin = -1;
	while(topoFin < TAMVETOR){
		if(h_Topo[i] >= h_Min[i]){
			maior = h_Pilha[h_Topo[i]];
			h_Topo[i]-=1;
		}else{
			maior = -1;
		}
		for(u=0; u < NUMTHREAD; u++){
			if(h_Topo[u] >= h_Min[u]){
				temp = h_Pilha[h_Topo[u]];
				h_Topo[u]-=1;
				if(maior < temp){
					if(maior != -1){
						h_Topo[i]+=1;
						h_Pilha[h_Topo[i]] = maior;
					}
					maior = temp;
				}else{
					h_Topo[u]+=1;
					h_Pilha[h_Topo[u]] = temp;
				}
			}
		}
		if(maior > -1){
			topoFin+=1;
			pilhaFinal[topoFin] = maior;
		}
	}
	
	tempo2();

	//mostrar(pilhaFinal, TAMVETOR);

	tempoFinal("mili segundos", "", NULL);

    	hipFree(d_Pilha);
    	hipFree(d_PilhaAux0);
    	hipFree(d_PilhaAux1);
	free(pilhaFinal);
	free(h_Topo);
	free(h_Min);
}

void mostrar(int *p, int capa){
	int t;	
	for(t = 0; t < capa; t++){
		printf("\tpos-%d:\t%d\n",t , p[t]);
	}
}
